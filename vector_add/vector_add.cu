#define N 10000000

#include <hip/hip_runtime.h>
#include<vector>
#include<iostream>
#include<assert.h>
#define MAX_ERR 1e-6
__global__ void vector_add(float *out, float *a, float *b, int n) {
    
    int stride=blockDim.x;//スレッドの全体の数
    int index=threadIdx.x;//現在のスレッドの番号(0=255)
    //もしsスレッドの数が一つならばindex=0,stride=1になる
    for(int i =index; i < N; i+=stride){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 
    float *d_a,*d_b,*d_out;
    // 1.Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

     //vectorの初期化
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    //a のデバイス用メモリの割り当て
    hipMalloc((void**)&d_a,sizeof(float)*N);
    hipMalloc((void**)&d_b,sizeof(float)*N);
    hipMalloc((void**)&d_out,sizeof(float)*N);
    //データをホストからデバイスに転送
    hipMemcpy(d_a,a,sizeof(float)*N,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,sizeof(float)*N,hipMemcpyHostToDevice);
    
    // Main function
    vector_add<<<1,128>>> (d_out,d_a, d_b, N);

    // hostメモリへデータを転送する
    hipMemcpy(out,d_out,sizeof(float)*N,hipMemcpyDeviceToHost);

    //Verification
    for(int i=0;i<N;i++){
        assert(fabs(out[i]-a[i]-b[i])<MAX_ERR);
    }

    printf("Assertion finished passed\n");

    //デバイスメモリの開放
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    //ホストメモリの開放
    free(a);
    free(b);
    free(out);
}
